#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <vector>
#include <sstream>
#include "nvToolsExt.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include "f_eval2.cuh"

using namespace std;

// x - m-dimensional point
// n - total number of points
__global__ void jacobian(double *deviceInput, double *deviceResult, const int m, const int n, const double h, const double epsilon)
{

    // number of blocks requried to process each point
    int blocksPerPoint = (m + 1023) / 1024;
    
    // idx in global deviceInput to start copying data from into shared memory
    int startBlockIdx = (blockIdx.x / blocksPerPoint) * m;

    // idx in global deviceInput till where to copy data into shared memory
    int endBlockIdx = startBlockIdx + m;

    //  we have to copy from deviceInput[startBlockIdx] to deviceInput[endBlockIdx].
    int elementsToCopyPerThread = (m + 1023) / 1024;

    int threadStartCopyIdx = startBlockIdx + (threadIdx.x * elementsToCopyPerThread);

    int threadEndCopyIdx = threadStartCopyIdx + elementsToCopyPerThread;

    int i = threadStartCopyIdx;
    int j = (threadIdx.x * elementsToCopyPerThread);

    extern __shared__ double sharedX[];
    while(i < threadEndCopyIdx && i < endBlockIdx) {
        sharedX[j] = deviceInput[i];
        i++; j++;
    }

    __syncthreads();

    int currentThread = ((blockIdx.x % blocksPerPoint) * blockDim.x) + threadIdx.x;

    if(currentThread < m) {

        double val1 = f_eval(sharedX, m, currentThread, h);

        double val2 = f_eval(sharedX, m, currentThread, -h);

        double result = (val1 - val2) / (2*epsilon);

        deviceResult[startBlockIdx + currentThread] = result;
    }
	
}

int main(int argc, char* argv[]) {
    string ipFile(argv[1]);
    string opFile(argv[2]);
    double epsilon = atof(argv[3]);

    const double h = 1e-2;

    int m, n;
    string line;

    ifstream ifile;
    ifile.open(ipFile);

    getline(ifile, line);
    n = stoi(line);

    getline(ifile, line);
    m = stoi(line);

    int nElements = m * n;
    double *hostInput = new double[m * n];
    double *hostResult = new double[m * n];

    // Read all the input points from the file
    for (int i = 0; getline(ifile, line);) {
        string number;
        stringstream s(line);

        while(getline(s, number, ',')) {
            hostInput[i] = stod(number);
            i++;
        }
    }

    ifile.close();
 
    // Setting up timing
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEventRecord(startEvent, 0);
   
    double *deviceInput, *deviceResult;

    hipError_t error;
    error = hipMalloc((void**) &deviceInput, nElements * sizeof(double));
    if (error != hipSuccess)
    {
         printf("hipMalloc returned error code %d: %s, line(%d)\n", error, hipGetErrorString(error), __LINE__);
         exit(EXIT_FAILURE);
    }

    error = hipMalloc((void**) &deviceResult, nElements * sizeof(double));
    if (error != hipSuccess)
    {
         printf("hipMalloc returned error code %d: %s, line(%d)\n", error, hipGetErrorString(error),  __LINE__);
         exit(EXIT_FAILURE);
    }

    // copy stuff
    hipMemcpy(deviceInput, hostInput, nElements * sizeof(double), hipMemcpyHostToDevice);

    int blocksPerPoint = (m + 1023) / 1024;
    int totalBlocks = n * blocksPerPoint;
    int threadsPerBlock = min(m, 1024);

    // call kernel
    jacobian<<<totalBlocks, threadsPerBlock, m*sizeof(double)>>>(deviceInput, deviceResult, m, n, h, epsilon);

    // copy back
    hipMemcpy(hostResult, deviceResult, nElements * sizeof(double), hipMemcpyDeviceToHost);    
   
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    // write to file
    ofstream ofile;
    ofile.open(opFile);

//    if( (m == 4096) && (n == 16384) ){
       for(int i = 0; i < n; i++) {
           for(int j = 0; j < m; j++) {

               if(j != m-1)
                   ofile << fixed << setprecision(6) << hostResult[i*m + j] << ",";
               else
                   ofile << fixed << setprecision(6) << hostResult[i*m + j] << endl;
           }
       }
//    }
    ofile.close();

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
  
    // Create a new file to log the execution time
    ofstream ologfile;
    ologfile.open("cudaShared2Log", ios_base::app);
    ologfile << n << " " <<  m << " " << fixed << setprecision(6) << elapsedTime << endl;
    ologfile.close();

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipFree(deviceInput);
    hipFree(deviceResult);
    delete[] hostInput, hostResult;

    return 0;
}
